#include "hip/hip_runtime.h"
#include "../include/02_gpu_conv2d_constMem.cuh"

__global__ void gpu_conv2d_constMem_kernel(float *d_N_ptr, float *d_P_ptr, int n_rows, int n_cols)
{
    // Which output element this thread works on
    int out_col = blockIdx.x*blockDim.x + threadIdx.x;
    int out_row = blockIdx.y*blockDim.y + threadIdx.y;
    
    // Check if output element is valid
    if (out_row < n_rows && out_col < n_cols) 
    {
        // Result (in thread register)
        float p_val = 0.0f;
        
        // Loop over elements of the filter array
        for (int f_row = 0; f_row < 2*FILTER_RADIUS+1; f_row++) 
        {
            for (int f_col = 0; f_col < 2*FILTER_RADIUS+1; f_col++) 
            {
                // Input element to filter element mapping
                int in_row = out_row + (f_row - FILTER_RADIUS);
                int in_col = out_col + (f_col - FILTER_RADIUS);
                
                // Boundary check
                if (in_row >= 0 && in_row < n_rows && in_col >= 0 && in_col < n_cols) 
                    p_val += d_F[f_row*(2*FILTER_RADIUS+1)+f_col] * d_N_ptr[in_row*n_cols + in_col];
            }
        }
        d_P_ptr[out_row*n_cols + out_col] = p_val;
    }
}